
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

// int computeConvolutionValue(int rowIndex, int colIndex, int mRows, int mCols, int mWidth, int convRows, int convCols, int m[mRows][mCols][mWidth], int c[convRows][convCols][mWidth]){
//     int res = 0;
//     for(int i=0; i<mWidth; i++){
//         for(int j=0; j<convCols; j++){
//             for(int k=0; k<convRows; k++){
//                 res += m[rowIndex+k][colIndex+j][i]*c[k][j][i];
//             }
//         }
//     }
//     return res;
// }

__global__ void computeConvolutionValue(int mRows, int mCols, int mWidth, int convRows, int convCols, int *m, int *c, int *output, int outputCols, int outputRows){
    int rowIndex=threadIdx.x, colIndex=blockIdx.x;
    // we need the index of m[rowIndex][colIndex][0]
    int res = 0;
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                //m[rowIndex+k][colIndex+j][i]*c[k][j][i]
                int indexOfC = i*(convCols*convRows) + (j*convRows) + k;
                int indexOfM = i*(mRows*mCols) + ((colIndex+j)*mRows) + (rowIndex+k);
                res += (c[indexOfC] * m[indexOfM]);
            }
        }
    }
    
    int indexToFillInValue = rowIndex + (outputRows*colIndex);
    output[indexToFillInValue] = res;
}

int main(){
    FILE *filePointer;
    char line[100] = {0};
    int mRows, mCols, mWidth, convRows, convCols;
    int outputRows, outputCols;
    int i=0;
    
    filePointer = fopen("input.txt", "r");
    while(!feof(filePointer)){
        fscanf(filePointer, "%d %d %d %d %d", &mRows, &mCols, &mWidth, &convRows, &convCols);
        break;
    }
//     int m[mRows][mCols][mWidth], c[convRows][convCols][mWidth];
    int *m, *c, *output;
    hipMallocManaged(&m, sizeof(int)*mRows*mCols*mWidth);
    hipMallocManaged(&c, sizeof(int)*convRows*convCols*mWidth);

    outputRows = mRows - convRows + 1;
    outputCols = mCols - convCols + 1;
//     int output[outputRows][outputCols];
    hipMallocManaged(&output, sizeof(int)*outputRows*outputCols);
    
    int m_input[mRows][mCols][mWidth], c_input[convRows][convCols][mWidth];
    
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<mCols; j++){
            for(int k=0; k<mRows; k++){
                while(!feof(filePointer)){
                    fscanf(filePointer, "%d", &m_input[k][j][i]);
                    break;
                }
            }
        }
    }
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                while(!feof(filePointer)){
                    fscanf(filePointer, "%d", &c_input[k][j][i]);
                    break;
                }
            }
        }
    }
    int val = 0;
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                c[val] = c_input[k][j][i];
                val++;
            }
        }
    }
    
    val = 0;
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<mCols; j++){
            for(int k=0; k<mRows; k++){
                m[val] = m_input[k][j][i];
                val++;
            }
        }
    }
    
    
    int m2[mRows][mCols][mWidth], c2[convRows][convCols][mWidth];
    
    for(int i=0; i<mWidth*convCols*convRows; i++){
        
        int z = i/(convCols*convRows);
        int v = i - z*(convCols*convRows);
        int y = v/convRows;
        v -= y*convRows;
        int x = v;
        c2[x][y][z] = c[i];
    }
    
    for(int i=0; i<mWidth*mCols*mRows; i++){
        
        int z = i/(mCols*mRows);
        int v = i - z*(mCols*mRows);
        int y = v/mRows;
        v -= y*mRows;
        int x = v;
        m2[x][y][z] = m[i];
    }
    
    computeConvolutionValue<<<outputCols, outputRows>>>(mRows, mCols, mWidth, convRows, convCols, m, c, output, outputCols, outputRows);
    hipDeviceSynchronize();
    
    int output_final[outputRows][outputCols];
    
    filePointer = fopen("output.txt", "w");
    
    val = 0;
    for(int i=0; i<outputCols*outputRows; i++){
//         printf("%d ", output[i]);
        fprintf(filePointer, "%d ", output[i]);
        val++;
        if(val==outputRows){
            val = 0;
//             printf("\n");
            fprintf(filePointer, "\n");
        }
    }
    
    fclose(filePointer);
    
    hipFree(m);
    hipFree(c);
    hipFree(output);
    
    return 0;
}