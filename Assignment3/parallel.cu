
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

// int computeConvolutionValue(int rowIndex, int colIndex, int mRows, int mCols, int mWidth, int convRows, int convCols, int m[mRows][mCols][mWidth], int c[convRows][convCols][mWidth]){
//     int res = 0;
//     for(int i=0; i<mWidth; i++){
//         for(int j=0; j<convCols; j++){
//             for(int k=0; k<convRows; k++){
//                 res += m[rowIndex+k][colIndex+j][i]*c[k][j][i];
//             }
//         }
//     }
//     return res;
// }

int main(){
    FILE *filePointer;
    char line[100] = {0};
    int mRows, mCols, mWidth, convRows, convCols;
    int outputRows, outputCols;
    int i=0;
    
    filePointer = fopen("input.txt", "r");
    while(!feof(filePointer)){
        fscanf(filePointer, "%d %d %d %d %d", &mRows, &mCols, &mWidth, &convRows, &convCols);
        break;
    }
//     int m[mRows][mCols][mWidth], c[convRows][convCols][mWidth];
    int *m, *c, *output;
    hipMallocManaged(&m, sizeof(int)*mRows*mCols*mWidth);
    hipMallocManaged(&c, sizeof(int)*convRows*convCols*mWidth);

    outputRows = mRows - convRows + 1;
    outputCols = mCols - convCols + 1;
//     int output[outputRows][outputCols];
    hipMallocManaged(&output, sizeof(int)*outputRows*outputCols);
    
    int m_input[mRows][mCols][mWidth], c_input[convRows][convCols][mWidth];

    // culprit
//     for(int i=0; i<mWidth; i++){
//         for(int j=0; j<mCols; j++){
//             for(int k=0; k<mRows; k++){
//                 while(!feof(filePointer)){
//                     int theIndexToPutIn;
//                     theIndexToPutIn = (i*(mRows)*(mCols)) + (j*(mRows)) + k;
//                     fscanf(filePointer, "%d", &m[theIndexToPutIn]);
//                     break;
//                 }
//             }
//         }
//     }
    
    
     // culprit
//     for(int i=0; i<mWidth; i++){
//         for(int j=0; j<convCols; j++){
//             for(int k=0; k<convRows; k++){
//                 while(!feof(filePointer)){
//                     int theIndexToPutIn;
//                     theIndexToPutIn = (i*(convRows)*(convCols)) + (j*(convRows)) + k;
//                     fscanf(filePointer, "%d", &c[theIndexToPutIn]);
//                     break;
//                 }
//             }
//         }
//     }
    
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<mCols; j++){
            for(int k=0; k<mRows; k++){
                while(!feof(filePointer)){
                    fscanf(filePointer, "%d", &m_input[k][j][i]);
                    break;
                }
            }
        }
    }
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                while(!feof(filePointer)){
                    fscanf(filePointer, "%d", &c_input[k][j][i]);
                    break;
                }
            }
        }
    }
    int val = 0;
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                c[val] = c_input[k][j][i];
                val++;
            }
        }
    }
    
    val = 0;
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<mCols; j++){
            for(int k=0; k<mRows; k++){
                m[val] = m_input[k][j][i];
                val++;
            }
        }
    }
    
    
    int m2[mRows][mCols][mWidth], c2[convRows][convCols][mWidth];
    
    for(int i=0; i<mWidth*convCols*convRows; i++){
        
        int z = i/(convCols*convRows);
        int v = i - z*(convCols*convRows);
        int y = v/convRows;
        v -= y*convRows;
        int x = v;
        c2[x][y][z] = c[i];
    }
    
    for(int i=0; i<mWidth*mCols*mRows; i++){
        
        int z = i/(mCols*mRows);
        int v = i - z*(mCols*mRows);
        int y = v/mRows;
        v -= y*mRows;
        int x = v;
        m2[x][y][z] = m[i];
    }
    
//     for(int i=0; i<mWidth*mCols*mRows; i++){
//         int id = i;
//         int z = id / ((mCols)*(mRows));
//         id -= (z * (mCols)*(mRows));
//         int y = id / (mRows);
//         int x = id % (mRows);
//         m2[x][y][z] = m[id];
//     }
    
//     int theZ = 0;
//     for(int i=0; i<mWidth*convCols*convRows; i++){
//         int id = i;
//         int z = id / ((convCols)*(convRows));
//         id -= (z * (convCols)*(convRows));
//         int y = id / (convRows);
//         int x = id % (convRows);
//         c2[x][y][z] = c[id];
//     }
    
//     for(int i=0; i<mWidth*convCols*convRows; i++){
//         printf("%d\n", c[i]);
//     }
    
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<mCols; j++){
            for(int k=0; k<mRows; k++){
                printf("%d ", m2[k][j][i]);
            }
            printf("\n");
        }
    }
    
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                printf("%d ", c2[k][j][i]);
            }
            printf("\n");
        }
    }
    
    
    
    return 0;
    
    

//     // start finding convolutions
//     for(int i=0; i<outputCols; i++){
//         for(int j=0; j<outputRows; j++){
//             output[j][i] = computeConvolutionValue(j, i, mRows, mCols, mWidth, convRows, convCols,  m, c);
//         }
//     }

//     for(int i=0; i<outputCols; i++){
//         for(int j=0; j<outputRows; j++){
//             printf("%d ", output[j][i]);
//         }
//         printf("\n");
//     }

//     filePointer = fopen("output.txt", "w");
//     for(int i=0; i<outputCols; i++){
//         for(int j=0; j<outputRows; j++){
//             fprintf(filePointer, "%d ", output[j][i]);
//         }
//         fprintf(filePointer, "\n");
//     }
//     fclose(filePointer);
    
    hipFree(m);
    hipFree(c);
    hipFree(output);
}