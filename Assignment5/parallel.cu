
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>
#include<math.h>

__global__ void parallelBubbleSorting(int* arr, int N, int maxElementsInArray){
    int threadIndex = threadIdx.x;
    int indexToStartFrom = threadIndex * maxElementsInArray;
    int indexTillEnd;
    if((threadIndex+1)*maxElementsInArray -1 >= N){
        indexTillEnd = N-1;
    }
    else{
        indexTillEnd = (threadIndex+1)*maxElementsInArray -1;
    }
    printf("%d %d\n", indexToStartFrom, indexTillEnd);
    // Can have any simple sorting algorithm here...
    /* Iterative Bubble Sort Algorithm */
    
    // bubble sort is crossing its limit with other arrays
    
    for(int i=indexToStartFrom; i<=indexTillEnd; i++){
        for(int j=indexToStartFrom; j<indexTillEnd; j++){
            if(arr[j]>arr[j+1]){
                int temp = arr[j];
                arr[j] = arr[j+1];
                arr[j+1] = temp;
            }
        }
    }
    /* Iterative Bubble Sort Algorithm End */
}

__global__ void parallelMerging(int* arr, int* arr2, int N, int maxElementsInArray){
    int threadIndex = threadIdx.x;
    int indexToStartFrom = threadIndex * maxElementsInArray;
    int indexTillEnd;
    if((threadIndex+1)*maxElementsInArray -1 >= N){
        indexTillEnd = N-1;
    }
    else{
        indexTillEnd = (threadIndex+1)*maxElementsInArray -1;
    }
    
//     printf("%d\n", indexTillEnd);
    
    
    int startIndexOfArray1=indexToStartFrom, startIndexOfArray2=((indexTillEnd+indexToStartFrom)/2)+1;
    int endIndexOfArray1=(indexTillEnd+indexToStartFrom)/2, endIndexOfArray2=indexTillEnd;
    int currentIndexOfArray1=startIndexOfArray1, currentIndexOfArray2=startIndexOfArray2;
    
    int indexTillWhereArrayIsFilled = indexToStartFrom;
  while (currentIndexOfArray1 <= endIndexOfArray1 &&
         currentIndexOfArray2 <= endIndexOfArray2) {
    if (arr[currentIndexOfArray1] < arr[currentIndexOfArray2]) {
      arr2[indexTillWhereArrayIsFilled] = arr[currentIndexOfArray1];
      currentIndexOfArray1++;
    } else {
      arr2[indexTillWhereArrayIsFilled] = arr[currentIndexOfArray2];
      currentIndexOfArray2++;
    }
    indexTillWhereArrayIsFilled++;
  }
    
    while(currentIndexOfArray1<=endIndexOfArray1){
        arr2[indexTillWhereArrayIsFilled] = arr[currentIndexOfArray1];
        currentIndexOfArray1++;
        indexTillWhereArrayIsFilled++;
    }
    while(currentIndexOfArray2<=endIndexOfArray2){
        arr2[indexTillWhereArrayIsFilled] = arr[currentIndexOfArray2];
        currentIndexOfArray2++;
        indexTillWhereArrayIsFilled++;
    }
    
    for(int i=indexToStartFrom; i<=indexTillEnd; i++){
        arr[i] = arr2[i];
    }
}

int main() {
  FILE *filePointer;
  char line[100] = {0};
  int N = 100000;
  int i=0;
    
  size_t size = N * sizeof(int);
  int *arr;
  int *arr2;
  hipMallocManaged(&arr, size);
  hipMallocManaged(&arr2, size);
  filePointer = fopen("input.txt", "r");

  while (fgets(line, sizeof(line), filePointer)) {
    arr[i] = atoi(line);
    i++;
  }
  fclose(filePointer);
    
    
  // strategy -> parllelly divide into really small blocks and bubble-sort them serially
  // merge them at the end
    
  // 5 Logarithmic Partitions
  int logartithmicPartitions = 5;
  
  // parallel execution in 32 threads to sync
  int threadCount = pow(2, logartithmicPartitions);
  int maxElementsInArray = N/threadCount;

  if(N%threadCount>0){
    maxElementsInArray++;
  }
    
  parallelBubbleSorting<<<1, threadCount>>>(arr, N, maxElementsInArray);
  hipDeviceSynchronize();
  
  for(int i=threadCount/2; i>=1; i/=2){
      maxElementsInArray *= 2;
      parallelMerging<<<1, i>>>(arr, arr2, N, maxElementsInArray);
      hipDeviceSynchronize();
  }
    
  for(int i=0; i<N; i++){
      if(i%(2*maxElementsInArray) == 0){
          printf("\n");
      }
    printf("%d\n", arr[i]);
  }
    
  return 0;
    
    
    
    
    
//   mergeSort(arr, 0, N - 1);
  
  filePointer = fopen("output.txt", "w");
  for(int i=0; i<N; i++)
  {
    fprintf(filePointer, "%d\n", arr[i]);
    //printf("%d\n", arr[i]);
  }
  hipDeviceSynchronize();
  hipFree(&arr);
  hipFree(&arr2);
}