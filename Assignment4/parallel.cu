
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

__global__ void computeSparseMultiplication(double *m_m, double *c_m, double *output_m, int N, int mCols, int mRows){
  int i;
  i = threadIdx.x;
  if (i < N)
  {
      double res = 0;
      for(int j=0; j<mCols; j++){
          int mIndex = i*mCols + j;
        res += m_m[mIndex]*c_m[j];
      }
      output_m[i] = res;
  }
}

int main(){
    FILE *filePointer;
    char line[100] = {0};
    int mRows, mCols, numberOfNonZeroElements;
    // number of elements in vector has to be equal to 
    
    filePointer = fopen("input.txt", "r");
    while(!feof(filePointer)){
        fscanf(filePointer, "%d %d %d", &mRows, &mCols, &numberOfNonZeroElements);
        break;
    }
    
    double m[mRows][mCols], c[mCols], output[mRows];

    
    for(int j=0; j<mCols; j++){
        for(int k=0; k<mRows; k++){
            m[k][j] = 0;
        }
        c[j] = 0;
    }
    
    for(int i=0; i<numberOfNonZeroElements; i++){
        while(!feof(filePointer)){
            int x, y;
            double val;
            fscanf(filePointer, "%d %d %lf", &x, &y, &val);
            m[x][y] = val;
            break;
        }
    }

    for(int i=0; i<mCols; i++){
        while(!feof(filePointer)){
            double val;
            fscanf(filePointer, "%lf", &val);
            c[i] = val;
            break;
        }
    }
    
    double *m_m, *c_m, *output_m;
    hipMallocManaged(&m_m, sizeof(double)*mRows*mCols);
    hipMallocManaged(&c_m, sizeof(double)*mCols);
    hipMallocManaged(&output_m, sizeof(double)*mRows);
    
     for(int j=0; j<mCols; j++){
        for(int k=0; k<mRows; k++){
            //m_m has to be 1D
            int id = j*mRows + k; 
            m_m[id] = m[k][j];
        }
        c_m[j] = c[j];
    }
    
    computeSparseMultiplication<<<1, mRows>>>(m_m, c_m, output_m, mRows, mCols, mRows);
//     printf("%s", cudaGetErrorString(cudaGetLastError()));
    hipDeviceSynchronize();
    
    filePointer = fopen("output.txt", "w");
    for(int i=0; i<mRows; i++){
        fprintf(filePointer, "%0.6lf\n", output_m[i]);
    }
    fclose(filePointer);
    
    hipFree(m_m);
    hipFree(c_m);
    hipFree(output_m);

}