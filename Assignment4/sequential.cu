
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

int computeConvolutionValue(int rowIndex, int colIndex, int mRows, int mCols, int mWidth, int convRows, int convCols, int m[mRows][mCols][mWidth], int c[convRows][convCols][mWidth]){
    int res = 0;
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                res += m[rowIndex+k][colIndex+j][i]*c[k][j][i];
            }
        }
    }
    return res;
}

int main(){
    FILE *filePointer;
    char line[100] = {0};
    int mRows, mCols, numberOfNonZeroElements;
    // number of elements in vector has to be equal to 
    
    filePointer = fopen("input.txt", "r");
    while(!feof(filePointer)){
        fscanf(filePointer, "%d %d %d", &mRows, &mCols, &numberOfNonZeroElements);
        break;
    }
    
    double m[mRows][mCols], c[mCols], output[mRows];

    
    for(int j=0; j<mCols; j++){
        for(int k=0; k<mRows; k++){
            m[k][j] = 0;
        }
        c[j] = 0;
    }

    for(int i=0; i<numberOfNonZeroElements; i++){
        while(!feof(filePointer)){
            int x, y;
            double val;
            fscanf(filePointer, "%d %d %lf", &x, &y, &val);
            m[x][y] = val;
            break;
        }
    }

    for(int i=0; i<mCols; i++){
        while(!feof(filePointer)){
            double val;
            fscanf(filePointer, "%lf", &val);
            c[i] = val;
            break;
        }
    }

    for(int i=0; i<mRows; i++){
        double res = 0;
        for(int j=0; j<mCols; j++){
            res += m[i][j]*c[j];
        }
        output[i] = res;
    }


    filePointer = fopen("output.txt", "w");
    for(int i=0; i<mRows; i++){
        fprintf(filePointer, "%0.6lf\n", output[i]);
    }
    fclose(filePointer);
}