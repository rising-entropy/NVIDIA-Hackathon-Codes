#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

__global__ void calculateDotProduct(double* a, double* b, double * c) 
{
	__shared__ double inter[threadsPerBlock];
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.x;
	
	float temp1 = 0;
	while (id < N){
		temp1 += a[id] * b[id];
		id += blockDim.x * gridDim.x;
	}
	
	inter[j] = temp1;
	__syncthreads();
	int b = blockDim.x/2;
	while (b != 0){
		if (j < i)
			inter[j] += inter[j+ b];
		__syncthreads();
		b /= 2;
	}
	
	if (j == 0)
		c[blockIdx.x] = inter[0];
}

int main()
{
  FILE *filePointer;
  char line[100] = {0};
	int Nv;
  filePointer = fopen("input.txt", "r");
  while (fgets(line, sizeof(line), filePointer)){
    Nv = atoi(line);
    break;
  }
                  double  *a, *b, sum, *c;
    	//float   *dev_a, *dev_b, *dev_c;
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));
	//double A[Nv], B[Nv],p;
  int i=0;
  while (fgets(line, sizeof(line), filePointer)) {
    if(i<Nv){
      A[i] = strtod(line, NULL);
    }
    else{
      B[i-Nv] = strtod(line, NULL);
    }
    i++;
  }
  fclose(filePointer);

const int threadsPerBlock = 256;
const int blocksPerGrid = (Nv+threadsPerBlock-1) / threadsPerBlock);
calculateDotProduct<<<blocksPerGrid, threadsPerBlock>>>(a, b, c);
hipDeviceSynchronize();

  filePointer = fopen("output.txt", "w");
  fprintf(filePointer, "%0.6lf\n", p);
  fclose(filePointer);
	return 0;
}