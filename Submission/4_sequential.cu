
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

int main(){
    FILE *filePointer;
    char line[100] = {0};
    int mRows, mCols, numberOfNonZeroElements;
    // number of elements in vector has to be equal to 
    
    filePointer = fopen("input.txt", "r");
    while(!feof(filePointer)){
        fscanf(filePointer, "%d %d %d", &mRows, &mCols, &numberOfNonZeroElements);
        break;
    }
    
    double m[mRows][mCols], c[mCols], output[mRows];

    
    for(int j=0; j<mCols; j++){
        for(int k=0; k<mRows; k++){
            m[k][j] = 0;
        }
        c[j] = 0;
    }

    for(int i=0; i<numberOfNonZeroElements; i++){
        while(!feof(filePointer)){
            int x, y;
            double val;
            fscanf(filePointer, "%d %d %lf", &x, &y, &val);
            m[x][y] = val;
            break;
        }
    }

    for(int i=0; i<mCols; i++){
        while(!feof(filePointer)){
            double val;
            fscanf(filePointer, "%lf", &val);
            c[i] = val;
            break;
        }
    }

    for(int i=0; i<mRows; i++){
        double res = 0;
        for(int j=0; j<mCols; j++){
            res += m[i][j]*c[j];
        }
        output[i] = res;
    }


    filePointer = fopen("output.txt", "w");
    for(int i=0; i<mRows; i++){
        fprintf(filePointer, "%0.6lf\n", output[i]);
    }
    fclose(filePointer);
}