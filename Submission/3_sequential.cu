
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

// int computeConvolutionValue(int rowIndex, int colIndex, int mRows, int mCols, int mWidth, int convRows, int convCols, int m[mRows][mCols][mWidth], int c[convRows][convCols][mWidth]){
//     int res = 0;
//     for(int i=0; i<mWidth; i++){
//         for(int j=0; j<convCols; j++){
//             for(int k=0; k<convRows; k++){
//                 res += m[rowIndex+k][colIndex+j][i]*c[k][j][i];
//             }
//         }
//     }
//     return res;
// }

int main(){
    FILE *filePointer;
    char line[100] = {0};
    int mRows, mCols, mWidth, convRows, convCols;
    int outputRows, outputCols;
    int i=0;
    
    filePointer = fopen("input.txt", "r");
    while(!feof(filePointer)){
        fscanf(filePointer, "%d %d %d %d %d", &mRows, &mCols, &mWidth, &convRows, &convCols);
        break;
    }
    int m[mRows][mCols][mWidth], c[convRows][convCols][mWidth];
    
    outputRows = mRows - convRows + 1;
    outputCols = mCols - convCols + 1;
    int output[outputRows][outputCols];
    
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<mCols; j++){
            for(int k=0; k<mRows; k++){
                while(!feof(filePointer)){
                    fscanf(filePointer, "%d", &m[k][j][i]);
                    break;
                }
            }
        }
    }
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                while(!feof(filePointer)){
                    fscanf(filePointer, "%d", &c[k][j][i]);
                    break;
                }
            }
        }
    }

    // start finding convolutions
    for(int i=0; i<outputCols; i++){
        for(int j=0; j<outputRows; j++){
            
            
            int res = 0;
            for(int ii=0; ii<mWidth; ii++){
                for(int jj=0; jj<convCols; jj++){
                    for(int kk=0; kk<convRows; kk++){
                        res += m[j+kk][i+jj][ii]*c[kk][jj][ii];
                    }
                }
            }
            
            
            output[j][i] = res;
        }
    }

//     for(int i=0; i<outputCols; i++){
//         for(int j=0; j<outputRows; j++){
//             printf("%d ", output[j][i]);
//         }
//         printf("\n");
//     }

    filePointer = fopen("output.txt", "w");
    for(int i=0; i<outputCols; i++){
        for(int j=0; j<outputRows; j++){
            fprintf(filePointer, "%d ", output[j][i]);
        }
        fprintf(filePointer, "\n");
    }
    fclose(filePointer);
}