
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

__global__ void calculateDotProduct(double* A, double* B, int N, double* res, int maxElementsInArray)
{
    int threadIndex = threadIdx.x;
    int indexToStartFrom = threadIndex * maxElementsInArray;
    int indexTillEnd;
    if((threadIndex+1)*maxElementsInArray -1 >= N){
        // indexTillEnd = N-1; //check later
        indexTillEnd = N;
    }
    else{
        indexTillEnd = (threadIndex+1)*maxElementsInArray -1;
    }
    double sum = 0;
    for(int i=indexToStartFrom; i<=indexTillEnd; i++){
        sum += (A[i]*B[i]);
    }
    res[threadIndex] = sum;
}

int main()
{
  FILE *filePointer;
  char line[100] = {0};
    int Nv;
  filePointer = fopen("input.txt", "r");
  while (fgets(line, sizeof(line), filePointer)){
    Nv = atoi(line);
    break;
  }
    
    int numberOfThreads = 218;
    size_t size_res = numberOfThreads * sizeof(double);
    int maxElementsInArray = Nv/numberOfThreads;
    if(Nv%numberOfThreads>0){
        maxElementsInArray++;
    }
    
    
    double* A;
    double* B;
    double* res;
    size_t size = Nv * sizeof(double);
   hipMallocManaged(&A, size);
  hipMallocManaged(&B, size);
  hipMallocManaged(&res, size_res);  
  int i=0;
  while (fgets(line, sizeof(line), filePointer)) {
    if(i<Nv){
      A[i] = strtod(line, NULL);
    }
    else{
      B[i-Nv] = strtod(line, NULL);
    }
    i++;
  }
  fclose(filePointer);
    
    
  calculateDotProduct<<<1, numberOfThreads>>>(A, B, Nv, res, maxElementsInArray);
  hipDeviceSynchronize();
  
    double p=0;
    for(int i=0; i<numberOfThreads; i++){
        p += res[i];
    }
    
  filePointer = fopen("output.txt", "w");
  fprintf(filePointer, "%0.6lf\n", p);
  fclose(filePointer);
    
    hipFree(A);
    hipFree(B);
    hipFree(res);
    return 0;
}