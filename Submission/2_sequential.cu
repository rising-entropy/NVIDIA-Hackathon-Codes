
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

double calculateDotProduct(double A[], double B[], int nv)
{
	double p = 0;
	for (int i = 0; i < nv; i++)
		p = p + A[i] * B[i];
	return p;
}

int main()
{
  FILE *filePointer;
  char line[100] = {0};
	int Nv;
  filePointer = fopen("input.txt", "r");
  while (fgets(line, sizeof(line), filePointer)){
    Nv = atoi(line);
    break;
  }
	double A[Nv], B[Nv],p;
  int i=0;
  while (fgets(line, sizeof(line), filePointer)) {
    if(i<Nv){
      A[i] = strtod(line, NULL);
    }
    else{
      B[i-Nv] = strtod(line, NULL);
    }
    i++;
  }
  fclose(filePointer);
  p = calculateDotProduct(A, B, Nv);
  filePointer = fopen("output.txt", "w");
  fprintf(filePointer, "%0.6lf\n", p);
  fclose(filePointer);
	return 0;
}