
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <stdlib.h>
#include<math.h>

__global__ void computePrefixSum(double* arr, double *prefixArraySums, int maxElementsInArray, int N){
    int threadIndex = threadIdx.x;
    int indexToStartFrom = threadIndex * maxElementsInArray;
    int indexTillEnd;
    if((threadIndex+1)*maxElementsInArray -1 >= N){
        // indexTillEnd = N-1; //check later
        indexTillEnd = N;
    }
    else{
        indexTillEnd = (threadIndex+1)*maxElementsInArray -1;
    }
    
    double sum = 0;
    for(int i=indexToStartFrom; i<=indexTillEnd; i++){
        sum += arr[i];
        arr[i] = sum;
    }
    prefixArraySums[threadIndex] = sum;
}

__global__ void computeFinalSum(double* arr, double *prefixArraySums, int maxElementsInArray, int N){
    int threadIndex = threadIdx.x;
    int indexToStartFrom = threadIndex * maxElementsInArray;
    int indexTillEnd;
    if((threadIndex+1)*maxElementsInArray -1 >= N){
        indexTillEnd = N;
    }
    else{
        indexTillEnd = (threadIndex+1)*maxElementsInArray -1;
    }
    double numberToAdd = prefixArraySums[threadIndex];
    for(int i=indexToStartFrom; i<=indexTillEnd; i++){
        arr[i] += numberToAdd;
    }
}

int main()
{
    FILE *filePointer;
    char line[100] = {0};
    int N;

    filePointer = fopen("input.txt", "r");
    while (fgets(line, sizeof(line), filePointer)) {
      N = atoi(line);
      break;
    }
    
    // divide into 32 sub-arrays
    int totalNumberOfSubArrays = 360;
    double doubleType;
    
    
    size_t size = N * sizeof(double);
    size_t size2 = totalNumberOfSubArrays * sizeof(double);
    
    double *arr;
    hipMallocManaged(&arr, size);
    
    double *prefixArraySums;
    hipMallocManaged(&prefixArraySums, size2);
    int maxElementsInArray = N/totalNumberOfSubArrays;
    if(N%totalNumberOfSubArrays>0){
        maxElementsInArray++;
    }
    
    int i=0;
    while (fgets(line, sizeof(line), filePointer)) {
      arr[i] = strtod(line, NULL);
      i++;
    }
    fclose(filePointer);
    
    computePrefixSum<<<1, totalNumberOfSubArrays>>>(arr, prefixArraySums, maxElementsInArray, N);
    
    hipDeviceSynchronize();
    
    // prefix sum
    double prefSumArray = 0;
    for(int j=0; j<totalNumberOfSubArrays; j++){
        double temp = prefSumArray;
        prefSumArray += prefixArraySums[j];
        prefixArraySums[j] = temp;
    }
    
    computeFinalSum<<<1, totalNumberOfSubArrays>>>(arr, prefixArraySums, maxElementsInArray, N);
    hipDeviceSynchronize();
    
    // only to summon the output
    
    filePointer = fopen("output-p.txt", "w");
    for(int i=0; i<N; i++)
    {
      fprintf(filePointer, "%0.4lf\n", arr[i]);
    }
    fclose(filePointer);
    hipFree(arr);  
}